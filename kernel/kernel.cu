#include "hip/hip_runtime.h"
#include <stdio.h>

#define INPUT_DIM 100
#define FILTER_DIM 5
#define CONV_LAYER_SIZE 10
#define REGION_DIM 20
#define CONV_OUT_DIM (INPUT_DIM / FILTER_DIM)
#define OUT_LAYER_SIZE 10
#define OUT_NEURON_DIM (CONV_OUT_DIM * CONV_OUT_DIM * CONV_LAYER_SIZE)

extern "C" __global__ void convolution(const double input[INPUT_DIM][INPUT_DIM], const double conv_layer[CONV_LAYER_SIZE][FILTER_DIM][FILTER_DIM], double conv_output[CONV_LAYER_SIZE][CONV_OUT_DIM][CONV_OUT_DIM]) {
  int i = blockIdx.x * FILTER_DIM; 
  int j = blockIdx.y * FILTER_DIM;
  int z = threadIdx.x; 

  if (i + FILTER_DIM > INPUT_DIM || j + FILTER_DIM > INPUT_DIM || z >= CONV_LAYER_SIZE) {
    return;
  }

  double prod = 0;
  for (int x = 0; x < FILTER_DIM; x++) {
    for (int y = 0; y < FILTER_DIM; y++) {
      prod += (input[i + x][j + y] * conv_layer[z][x][y]);
    }
  }
  conv_output[z][i / FILTER_DIM][j / FILTER_DIM] = prod;
}

extern "C" __global__ void relu(double conv_output[CONV_LAYER_SIZE][CONV_OUT_DIM][CONV_OUT_DIM]) {
  int i = blockIdx.x; 
  int j = blockIdx.y;

  int z = threadIdx.x; 

  if (i >= CONV_OUT_DIM || j >= CONV_OUT_DIM || z >= CONV_LAYER_SIZE) {
    return;
  }

  if (conv_output[z][i][j] < 0) {
    conv_output[z][i][j] = 0.0;
  }
}

extern "C" __global__ void output(double conv_output[CONV_LAYER_SIZE][CONV_OUT_DIM][CONV_OUT_DIM], double weights[OUT_LAYER_SIZE][OUT_NEURON_DIM], double output[OUT_LAYER_SIZE]) {
  int z = threadIdx.x; 

  if (x >= CONV_OUT_DIM || y >= CONV_OUT_DIM || z >= OUT_LAYER_SIZE) {
    return;
  }

  double prod = 0;
  for (int i = 0; i < CONV_OUT_DIM; i++) {
    for (int j = 0; j < CONV_OUT_DIM; j++) {
      for (int k = 0; k < OUT_LAYER_SIZE; k++) {
        int id = k * 400 + i * 20 + j;
        prod += (conv_output[k][i][j] * weights[z][id]);
      }
    }
  }

  output[z] = prod;
}
